#include "hip/hip_runtime.h"
// fcm_mobility_solver.cu

#include <iomanip>
#include <iostream>
#include <cmath>
#include <algorithm>
#include "fcm_mobility_solver.hpp"
#include "../cuda_functions.hpp"

fcm_mobility_solver::~fcm_mobility_solver(){}

fcm_mobility_solver::fcm_mobility_solver(){
  Real values[100];
  std::string fcm_folder = "../CUFCM/";
  std::vector<std::string> datafile_names{3};
  read_config(values, datafile_names, "../CUFCM/simulation_info_long");
  for(int i = 0; i < 3; i++){
    datafile_names[i] = fcm_folder + datafile_names[i];
  }

  pars.N = NSWIM*NFIL*NSEG + NSWIM*NBLOB;
  pars.rh = values[1];
  pars.alpha = values[2];
  pars.beta = values[3];
  pars.eta = values[4];
  int npts = values[5];
  pars.nx = npts;
  pars.ny = npts;
  pars.nz = npts;
  pars.repeat = values[8];
  pars.prompt = values[9];
  pars.boxsize = values[13];
  // pars.boxsize = pars.rh/1.7724538509055159 * pars.nx / pars.alpha;

  cufcm_solver = new FCM_solver(pars);
  cufcm_solver->init_aux_for_filament();
}


void fcm_mobility_solver::free_host_memory(){
  
  delete[] num_segs;
  delete[] num_blobs;

  hipHostFree(v_segs_host);
  hipHostFree(v_blobs_host);
  hipHostFree(x_segs_host);
  hipHostFree(x_blobs_host);
  hipHostFree(f_segs_host);
  hipHostFree(f_blobs_host);
  hipHostFree(f_blobs_repulsion_host);

}

void fcm_mobility_solver::free_device_memory(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipFree(v_segs_device[n]);
    hipFree(v_blobs_device[n]);
    hipFree(x_segs_device[n]);
    hipFree(x_blobs_device[n]);
    hipFree(f_segs_device[n]);
    hipFree(f_blobs_device[n]);
    hipFree(f_blobs_repulsion_device[n]);

  }

  delete[] v_segs_device;
  delete[] v_blobs_device;
  delete[] x_segs_device;
  delete[] x_blobs_device;
  delete[] f_segs_device;
  delete[] f_blobs_device;
  delete[] f_blobs_repulsion_device;

}

void fcm_mobility_solver::allocate_host_memory(){
  
  std::cout << std::endl << std::endl << "Running on all GPUs visible to this shell environment, as defined by the environment variable CUDA_VISIBLE_DEVICES." << std::endl;

  hipGetDeviceCount(&num_gpus);

  std::cout <<  "Found " << num_gpus << " GPU(s)." << std::endl;

  hipSetDevice(0);

  // Allocate pinned host memory to allow async copying
  hipHostAlloc(&v_segs_host, 6*NSWIM*NFIL*NSEG*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&v_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&x_segs_host, 3*NSWIM*NFIL*NSEG*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&x_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&f_segs_host, 6*NSWIM*NFIL*NSEG*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&f_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&f_blobs_repulsion_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);

  num_segs = new int[num_gpus];
  num_blobs = new int[num_gpus];

  num_segs[0] = NSWIM*NFIL*NSEG;
  num_blobs[0] = NSWIM*NBLOB;

  for (int n = 1; n < num_gpus; n++){

    num_segs[n] = (NSWIM*NFIL*NSEG)/num_gpus;
    num_segs[0] -= num_segs[n];

    num_blobs[n] = (NSWIM*NBLOB)/num_gpus;
    num_blobs[0] -= num_blobs[n];

  }

}

void fcm_mobility_solver::allocate_device_memory(){
  
  v_segs_device = new Real*[num_gpus];
  v_blobs_device = new Real*[num_gpus];
  x_segs_device = new Real*[num_gpus];
  x_blobs_device = new Real*[num_gpus];
  f_segs_device = new Real*[num_gpus];
  f_blobs_device = new Real*[num_gpus];
  f_blobs_repulsion_device = new Real*[num_gpus];

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    hipMalloc(&v_segs_device[n], 6*num_segs[n]*sizeof(Real));
    hipMalloc(&v_blobs_device[n], 3*num_blobs[n]*sizeof(Real));

    hipMalloc(&x_segs_device[n], 3*NSWIM*NFIL*NSEG*sizeof(Real));
    hipMalloc(&x_blobs_device[n], 3*NSWIM*NBLOB*sizeof(Real));

    hipMalloc(&f_segs_device[n], 6*NSWIM*NFIL*NSEG*sizeof(Real));
    hipMalloc(&f_blobs_device[n], 3*NSWIM*NBLOB*sizeof(Real));
    hipMalloc(&f_blobs_repulsion_device[n], 3*num_blobs[n]*sizeof(Real));

  }

}

void fcm_mobility_solver::copy_segment_positions_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(x_segs_device[n], x_segs_host, 3*NSWIM*NFIL*NSEG*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void fcm_mobility_solver::copy_segment_forces_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(f_segs_device[n], f_segs_host, 6*NSWIM*NFIL*NSEG*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void fcm_mobility_solver::copy_blob_positions_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(x_blobs_device[n], x_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void fcm_mobility_solver::copy_blob_forces_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(f_blobs_device[n], f_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void fcm_mobility_solver::copy_interparticle_blob_forces_to_host(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(&f_blobs_repulsion_host[3*start_blob], f_blobs_repulsion_device[n], 3*num_blobs[n]*sizeof(Real), hipMemcpyDeviceToHost);
    start_blob += num_blobs[n];

  }

}

void fcm_mobility_solver::copy_blob_velocities_to_host(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(&v_blobs_host[3*start_blob], v_blobs_device[n], 3*num_blobs[n]*sizeof(Real), hipMemcpyDeviceToHost);
    start_blob += num_blobs[n];

  }

}

void fcm_mobility_solver::copy_segment_velocities_to_host(){

  int start_seg = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    hipMemcpyAsync(&v_segs_host[6*start_seg], v_segs_device[n], 6*num_segs[n]*sizeof(Real), hipMemcpyDeviceToHost);

    start_seg += num_segs[n];

  }

}

void fcm_mobility_solver::apply_interparticle_forces(){


  #if !PRESCRIBED_CILIA

    hipSetDevice(0);

    const int num_thread_blocks = (std::max<int>(num_segs[0], num_blobs[0]) + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    // FILE *pfile;

    // periodic_barrier_forces<<<num_thread_blocks, THREADS_PER_BLOCK>>>(
    //   f_segs_device[0], f_blobs_repulsion_device[0],
    //   x_segs_device[0], x_blobs_device[0],
    //   num_segs[0],
    //   num_blobs[0],
    //   pars.boxsize);

    // hipMemcpy(&f_blobs_repulsion_host[0], f_blobs_repulsion_device[0], 3*num_blobs[0]*sizeof(Real), hipMemcpyDeviceToHost);
    // pfile = fopen("barrier_force_fil.dat", "w");
    // for(int i = 0; i < num_blobs[0]; i++){
    //     fprintf(pfile, "FIL %d %.8f %.8f %.8f %.8f %.8f %.8f \n", 
    //     i, f_blobs_repulsion_host[3*i], f_blobs_repulsion_host[3*i+1], f_blobs_repulsion_host[3*i+2],
    //     x_blobs_host[3*i], x_blobs_host[3*i+1], x_blobs_host[3*i+2]);
    //     }
    // fprintf(pfile, "\n#");
    // fclose(pfile);

    cufcm_solver->reform_data(x_segs_device[0], f_segs_device[0], v_segs_device[0],
                            x_blobs_device[0], f_blobs_repulsion_device[0], v_blobs_device[0],
                            num_segs[0], num_blobs[0], true);

    cufcm_solver->apply_repulsion_for_timcode(num_segs[0], num_blobs[0]);

    cufcm_solver->reform_data_back(x_segs_device[0], f_segs_device[0], v_segs_device[0],
                                  x_blobs_device[0], f_blobs_repulsion_device[0], v_blobs_device[0],
                                  num_segs[0], num_blobs[0], true);
                                
    // hipMemcpy(&f_blobs_repulsion_host[0], f_blobs_repulsion_device[0], 3*num_blobs[0]*sizeof(Real), hipMemcpyDeviceToHost);
    // pfile = fopen("barrier_force_fcm.dat", "w");
    // for(int i = 0; i < num_blobs[0]; i++){
    //     fprintf(pfile, "FCM %d %.8f %.8f %.8f %.8f %.8f %.8f \n", 
    //     i, f_blobs_repulsion_host[3*i], f_blobs_repulsion_host[3*i+1], f_blobs_repulsion_host[3*i+2],
    //     x_blobs_host[3*i], x_blobs_host[3*i+1], x_blobs_host[3*i+2]);
    //     }
    // fprintf(pfile, "\n#");
    // fclose(pfile);

  #endif

}

void fcm_mobility_solver::wait_for_device(){

  for (int n = 0; n < num_gpus; n++){

      hipSetDevice(n);
      hipDeviceSynchronize();

    }

}

void fcm_mobility_solver::evaluate_segment_segment_mobility(){

    int start_seg = 0;

    hipSetDevice(0);

    int num_thread_blocks = (num_segs[0] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    // Mss_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_segs_device[0], f_segs_device[0], x_segs_device[0], start_seg, num_segs[0]);

    cufcm_solver->reform_data(x_segs_device[0], f_segs_device[0], v_segs_device[0],
                              x_blobs_device[0], f_blobs_device[0], v_blobs_device[0],
                              num_segs[0], num_blobs[0], false);
                    
    cufcm_solver->Mss();

    cufcm_solver->reform_data_back(x_segs_device[0], f_segs_device[0], v_segs_device[0],
                                  x_blobs_device[0], f_blobs_device[0], v_blobs_device[0],
                                  num_segs[0], num_blobs[0], false);

    start_seg += num_segs[0];


}

void fcm_mobility_solver::evaluate_segment_blob_mobility(){

  int start_seg = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    int num_thread_blocks = (num_segs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Msb_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_segs_device[n], f_blobs_device[n], x_segs_device[n], x_blobs_device[n], start_seg, num_segs[n]);

    start_seg += num_segs[n];

  }

}

void fcm_mobility_solver::evaluate_blob_blob_mobility(){

  hipSetDevice(0);

  int num_thread_blocks = (num_segs[0] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

  cufcm_solver->reform_data(x_segs_device[0], f_segs_device[0], v_segs_device[0],
                            x_blobs_device[0], f_blobs_device[0], v_blobs_device[0],
                            num_segs[0], num_blobs[0], false);

  cufcm_solver->Mss();

  cufcm_solver->reform_data_back(x_segs_device[0], f_segs_device[0], v_segs_device[0],
                                  x_blobs_device[0], f_blobs_device[0], v_blobs_device[0],
                                  num_segs[0], num_blobs[0], false);

}

void fcm_mobility_solver::evaluate_blob_segment_mobility(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    const int num_thread_blocks = (num_blobs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Mbs_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_blobs_device[n], f_segs_device[n], x_blobs_device[n], x_segs_device[n], start_blob, num_blobs[n]);

    start_blob += num_blobs[n];

  }

}

void fcm_mobility_solver::write_repulsion(){
  FILE *pfile;
  hipMemcpy(&f_blobs_repulsion_host[0], f_blobs_repulsion_device[0], 3*num_blobs[0]*sizeof(Real), hipMemcpyDeviceToHost);
  pfile = fopen("barrier_force_fil.dat", "w");
  for(int i = 0; i < num_blobs[0]; i++){
      fprintf(pfile, "FIL %d %.8f %.8f %.8f %.8f %.8f %.8f \n", 
      i, f_blobs_repulsion_host[3*i], f_blobs_repulsion_host[3*i+1], f_blobs_repulsion_host[3*i+2],
      x_blobs_host[3*i], x_blobs_host[3*i+1], x_blobs_host[3*i+2]);
      }
  fprintf(pfile, "\n#");
  fclose(pfile);
}

