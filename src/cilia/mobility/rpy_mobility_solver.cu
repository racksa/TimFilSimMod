#include "hip/hip_runtime.h"
// rpy_mobility_solver.cu

#include <iomanip>
#include <iostream>
#include <cmath>
#include <algorithm>
#include "rpy_mobility_solver.hpp"
#include "../cuda_functions.hpp"

rpy_mobility_solver::~rpy_mobility_solver(){}

rpy_mobility_solver::rpy_mobility_solver(){}

void rpy_mobility_solver::free_host_memory(){
  
  delete[] num_segs;
  delete[] num_blobs;

  hipHostFree(v_segs_host);
  hipHostFree(v_blobs_host);
  hipHostFree(x_segs_host);
  hipHostFree(x_blobs_host);
  hipHostFree(f_segs_host);
  hipHostFree(f_blobs_host);
  hipHostFree(f_blobs_repulsion_host);

}

void rpy_mobility_solver::free_device_memory(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipFree(v_segs_device[n]);
    hipFree(v_blobs_device[n]);
    hipFree(x_segs_device[n]);
    hipFree(x_blobs_device[n]);
    hipFree(f_segs_device[n]);
    hipFree(f_blobs_device[n]);
    hipFree(f_blobs_repulsion_device[n]);

  }

  delete[] v_segs_device;
  delete[] v_blobs_device;
  delete[] x_segs_device;
  delete[] x_blobs_device;
  delete[] f_segs_device;
  delete[] f_blobs_device;
  delete[] f_blobs_repulsion_device;

}

void rpy_mobility_solver::allocate_host_memory(){
  
  std::cout << std::endl << std::endl << "Running on all GPUs visible to this shell environment, as defined by the environment variable CUDA_VISIBLE_DEVICES." << std::endl;

  hipGetDeviceCount(&num_gpus);

  std::cout <<  "Found " << num_gpus << " GPU(s)." << std::endl;

  hipSetDevice(0);

  // Allocate pinned host memory to allow async copying
  hipHostAlloc(&v_segs_host, 6*NSWIM*NFIL*NSEG*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&v_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&x_segs_host, 3*NSWIM*NFIL*NSEG*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&x_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&f_segs_host, 6*NSWIM*NFIL*NSEG*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&f_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);
  hipHostAlloc(&f_blobs_repulsion_host, 3*NSWIM*NBLOB*sizeof(Real), hipHostMallocPortable);

  num_segs = new int[num_gpus];
  num_blobs = new int[num_gpus];

  num_segs[0] = NSWIM*NFIL*NSEG;
  num_blobs[0] = NSWIM*NBLOB;

  for (int n = 1; n < num_gpus; n++){

    num_segs[n] = (NSWIM*NFIL*NSEG)/num_gpus;
    num_segs[0] -= num_segs[n];

    num_blobs[n] = (NSWIM*NBLOB)/num_gpus;
    num_blobs[0] -= num_blobs[n];

  }

}

void rpy_mobility_solver::allocate_device_memory(){
  
  v_segs_device = new Real*[num_gpus];
  v_blobs_device = new Real*[num_gpus];
  x_segs_device = new Real*[num_gpus];
  x_blobs_device = new Real*[num_gpus];
  f_segs_device = new Real*[num_gpus];
  f_blobs_device = new Real*[num_gpus];
  f_blobs_repulsion_device = new Real*[num_gpus];

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    hipMalloc(&v_segs_device[n], 6*num_segs[n]*sizeof(Real));
    hipMalloc(&v_blobs_device[n], 3*num_blobs[n]*sizeof(Real));

    hipMalloc(&x_segs_device[n], 3*NSWIM*NFIL*NSEG*sizeof(Real));
    hipMalloc(&x_blobs_device[n], 3*NSWIM*NBLOB*sizeof(Real));

    hipMalloc(&f_segs_device[n], 6*NSWIM*NFIL*NSEG*sizeof(Real));
    hipMalloc(&f_blobs_device[n], 3*NSWIM*NBLOB*sizeof(Real));
    hipMalloc(&f_blobs_repulsion_device[n], 3*num_blobs[n]*sizeof(Real));

  }

}

void rpy_mobility_solver::copy_segment_positions_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(x_segs_device[n], x_segs_host, 3*NSWIM*NFIL*NSEG*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void rpy_mobility_solver::copy_segment_forces_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(f_segs_device[n], f_segs_host, 6*NSWIM*NFIL*NSEG*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void rpy_mobility_solver::copy_blob_positions_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(x_blobs_device[n], x_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void rpy_mobility_solver::copy_blob_forces_to_device(){

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(f_blobs_device[n], f_blobs_host, 3*NSWIM*NBLOB*sizeof(Real), hipMemcpyHostToDevice);

  }

}

void rpy_mobility_solver::copy_interparticle_blob_forces_to_host(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(&f_blobs_repulsion_host[3*start_blob], f_blobs_repulsion_device[n], 3*num_blobs[n]*sizeof(Real), hipMemcpyDeviceToHost);
    start_blob += num_blobs[n];

  }

}

void rpy_mobility_solver::copy_blob_velocities_to_host(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);
    hipMemcpyAsync(&v_blobs_host[3*start_blob], v_blobs_device[n], 3*num_blobs[n]*sizeof(Real), hipMemcpyDeviceToHost);
    start_blob += num_blobs[n];

  }

}

void rpy_mobility_solver::copy_segment_velocities_to_host(){

  int start_seg = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    hipMemcpyAsync(&v_segs_host[6*start_seg], v_segs_device[n], 6*num_segs[n]*sizeof(Real), hipMemcpyDeviceToHost);

    start_seg += num_segs[n];

  }

}

void rpy_mobility_solver::apply_interparticle_forces(){

  #if !PRESCRIBED_CILIA

    int start_seg = 0;
    int start_blob = 0;

    for (int n = 0; n < num_gpus; n++){

      hipSetDevice(n);

      const int num_thread_blocks = (std::max<int>(num_segs[n], num_blobs[n]) + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
      
      barrier_forces<<<num_thread_blocks, THREADS_PER_BLOCK>>>(f_segs_device[n], f_blobs_repulsion_device[n], x_segs_device[n], x_blobs_device[n], start_seg, num_segs[n], start_blob, num_blobs[n]);

      if (num_gpus > 1){

        // In the barrier_forces(...) kernel, each GPU only evaluates barrier forces for the same subset of segments it evaluates velocities for.
        // So if we have multiple GPUs but don't copy back the forces to share, HIs between segments will only include any barrier force contributions
        // if their velocities happen to be computed on the same GPU.
        hipMemcpyAsync(&f_segs_host[6*start_seg], &f_segs_device[n][6*start_seg], 6*num_segs[n]*sizeof(Real), hipMemcpyDeviceToHost);

      }

      start_seg += num_segs[n];
      start_blob += num_blobs[n];

    }

    if (num_gpus > 1){

      rpy_mobility_solver::wait_for_device();
      rpy_mobility_solver::copy_segment_forces_to_device();

    }

  #endif

}

void rpy_mobility_solver::wait_for_device(){

  for (int n = 0; n < num_gpus; n++){

      hipSetDevice(n);
      hipDeviceSynchronize();

    }

}

void rpy_mobility_solver::evaluate_segment_segment_mobility(){

  int start_seg = 0;

    for (int n = 0; n < num_gpus; n++){

      hipSetDevice(n);

      int num_thread_blocks = (num_segs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

      Mss_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_segs_device[n], f_segs_device[n], x_segs_device[n], start_seg, num_segs[n]);

      start_seg += num_segs[n];

    }

}

void rpy_mobility_solver::evaluate_segment_blob_mobility(){

  int start_seg = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    int num_thread_blocks = (num_segs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Msb_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_segs_device[n], f_blobs_device[n], x_segs_device[n], x_blobs_device[n], start_seg, num_segs[n]);

    start_seg += num_segs[n];

  }

}

void rpy_mobility_solver::evaluate_blob_blob_mobility(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    int num_thread_blocks = (num_blobs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Mbb_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_blobs_device[n], f_blobs_device[n], x_blobs_device[n], start_blob, num_blobs[n]);

    start_blob += num_blobs[n];

  }

}

void rpy_mobility_solver::evaluate_blob_segment_mobility(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    const int num_thread_blocks = (num_blobs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Mbs_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_blobs_device[n], f_segs_device[n], x_blobs_device[n], x_segs_device[n], start_blob, num_blobs[n]);

    start_blob += num_blobs[n];

  }

}

void rpy_mobility_solver::evaluate_full_mobility(){
  evaluate_segment_segment_mobility();
  evaluate_segment_blob_mobility();
  copy_segment_velocities_to_host();

  evaluate_blob_blob_mobility();
  evaluate_blob_segment_mobility();
  copy_blob_velocities_to_host();
}